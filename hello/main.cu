#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void hello_cuda(){
    printf("Hello CUDA world \n");
}

int main(){
    hello_cuda<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
